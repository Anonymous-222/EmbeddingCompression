#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void robe_lookup_kernel(const float *input, const int *ids, const int *x,
                                        float *output, size_t size,
                                        size_t length, int Bg, int Cg, int Dg, size_t roarsz) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size)
        return;
    int id = ids[index];
    float *output_ptr = output + length * index;
    if (id < 0 || id >= roarsz) {
        for (int i = 0; i < length; i++)
            output_ptr[i] = 0;
    } else {
        //const float *input_ptr = input + id;
        int X = x[index];
        for (int i = 0; i < length; i++){
            int sgn = (((1ll * X * Bg + 1ll * i * Cg + Dg)%998244353+998244353)%998244353 %2)*2-1;
            output_ptr[i] = input[i+id<roarsz?(i+id):(i+id-roarsz)] * sgn;
        }
    }
}

int DLGpuRobeLookUp(const DLArrayHandle input, const DLArrayHandle ids, const DLArrayHandle x,
                         DLArrayHandle output, int len, int Bg, int Cg, int Dg,
                         DLStreamHandle stream_handle = NULL) {
    assert(input->ndim == 1);
    size_t size = 1;
    for (int i = 0; i < output->ndim; i++) {
        if (i < output->ndim - 1) {
            assert(ids->shape[i] == output->shape[i]);
        } else if (i == output->ndim - 1) {
            assert(len == output->shape[i]);
        }
    }
    for (int i = 0; i < ids->ndim; i++) {
        size = size * ids->shape[i];
    }
    size_t roarsz = input->shape[0];
    size_t length = (size_t)(len);
    dim3 blocks;
    dim3 threads;
    float *output_data = (float *)output->data;
    const float *input_data = (const float *)input->data;
    const int *id_list = (const int *)ids->data;
    const int *x_list = (const int *)x->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        robe_lookup_kernel<<<blocks, threads, 0,
                                  *(hipStream_t *)stream_handle->handle>>>(
            input_data, id_list, x_list, output_data, size, length, roarsz, Bg, Cg, Dg);
    else
        robe_lookup_kernel<<<blocks, threads>>>(
            input_data, id_list, x_list, output_data, size, length, roarsz, Bg, Cg, Dg);
    return 0;
}

__global__ void array_set_zero_kernel1(float *output, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = 0;
}


__global__ void robe_lookup_gradient_kernel(const float *output_grad_data,
                                                 const int *ids,
                                                 float *input_grad_data,
                                                 size_t size, size_t length) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size)
        return;
    int id = ids[index];
    const float *output_grad_ptr = output_grad_data + length * index;
    float *input_grad_ptr = input_grad_data + length * id;
    for (int i = 0; i < length; i++)
        atomicAdd(input_grad_ptr + i, *(output_grad_ptr + i));
}

int DLGpuRobeLookUp_Gradient(const DLArrayHandle output_grad,
                                  const DLArrayHandle ids,
                                  DLArrayHandle input_grad,
                                  DLStreamHandle stream_handle = NULL) {
    assert(input_grad->ndim == 2);
    size_t size = 1;
    for (int i = 0; i < output_grad->ndim; i++) {
        if (i < output_grad->ndim - 1) {
            assert(ids->shape[i] == output_grad->shape[i]);
        } else if (i == output_grad->ndim - 1) {
            assert(input_grad->shape[1] == output_grad->shape[i]);
        }
    }
    for (int i = 0; i < ids->ndim; i++) {
        size = size * ids->shape[i];
    }
    size_t length = input_grad->shape[1];
    dim3 blocks;
    dim3 threads;
    const float *output_grad_data = (const float *)output_grad->data;
    float *input_grad_data = (float *)input_grad->data;
    const int *id_list = (const int *)ids->data;

    size_t input_grad_size = 1;
    for (int i = 0; i < input_grad->ndim; i++) {
        input_grad_size *= input_grad->shape[i];
    }
    if (input_grad_size <= 1024) {
        threads.x = input_grad_size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (input_grad_size + 1023) / 1024;
    }
    if (stream_handle)
        array_set_zero_kernel1<<<blocks, threads, 0,
                                *(hipStream_t *)stream_handle->handle>>>(
            input_grad_data, input_grad_size);
    else
        array_set_zero_kernel1<<<blocks, threads>>>(input_grad_data,
                                                   input_grad_size);

    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        robe_lookup_gradient_kernel<<<
            blocks, threads, 0, *(hipStream_t *)stream_handle->handle>>>(
            output_grad_data, id_list, input_grad_data, size, length);
    else
        robe_lookup_gradient_kernel<<<blocks, threads>>>(
            output_grad_data, id_list, input_grad_data, size, length);
    return 0;
}