#include "hip/hip_runtime.h"
#include "gpu_runtime.h"


__global__ void sgd_robe_update(const float *grad_data,
                                  const int *indices_data, float *param_data,
                                  size_t size, size_t length, float lr, int roarsz) {
    size_t thread_ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_ind >= size)
        return;
    size_t index = thread_ind / length;
    size_t offset = thread_ind % length;
    int id = indices_data[index];
    if (id < 0)
        return;
    const float cur_grad = grad_data[thread_ind];
    atomicAdd(param_data + (id + offset<roarsz?id+offset:id+offset-roarsz), -lr * cur_grad);
//    param_data[(id + offset<roarsz?id+offset:id+offset-roarsz)] -= lr * cur_grad;
}

int SGDOptimizerRobeUpdate(DLArrayHandle param,
                             const DLArrayHandle grad_indices,
                             const DLArrayHandle grad_values, float lr,
                             DLStreamHandle stream_handle = NULL) {
    size_t size = ArrSize(grad_values);
    size_t roarsz = ArrSize(param);
    
    size_t length = (grad_values->shape[(grad_values->ndim) - 1]);

    const float *grad_data = (const float *)grad_values->data;
    float *param_data = (float *)param->data;
    const int *indices_data = (const int *)grad_indices->data;

    dim3 blocks;
    dim3 threads;
    ThreadBlock1D(threads, blocks, size);

    if (stream_handle)
        sgd_robe_update<<<blocks, threads, 0,
                            *(hipStream_t *)stream_handle->handle>>>(
            grad_data, indices_data, param_data, size, length, lr, roarsz);
    else
        sgd_robe_update<<<blocks, threads>>>(grad_data, indices_data,
                                               param_data, size, length, lr, roarsz);
    return 0;
}
