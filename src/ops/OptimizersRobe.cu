#include "hip/hip_runtime.h"
#include "gpu_runtime.h"


__global__ void sgd_robe_update(const float *grad_data,
                                  const int *indices_data, const int *x_data, float *param_data,
                                  size_t size, size_t length, float lr, int roarsz,
                                  int Bg, int Cg, int Dg,int Z,int blk,int MO) {
    size_t thread_ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_ind >= size)
        return;
    size_t index = thread_ind / length;
    size_t i = thread_ind % length;
    const float cur_grad = grad_data[thread_ind];

    const int X = x_data[index];
    int sgn = (((1ll * X * Bg + 1ll * i * Cg + Dg)%MO+MO)%MO %2)*2-1;
    
    
    int id = indices_data[index * blk+i/Z];
    if (id < 0)
        return;
    atomicAdd(param_data + ((id+i%Z)<roarsz?(id+i%Z):(id+i%Z)-roarsz), -lr * cur_grad * sgn);
//    param_data[(id + offset<roarsz?id+offset:id+offset-roarsz)] -= lr * cur_grad;
}

int SGDOptimizerRobeUpdate(DLArrayHandle param,
                             const DLArrayHandle grad_indices,
                             const DLArrayHandle grad_values,
                             const DLArrayHandle grad_x,
                             float lr, int Bg, int Cg, int Dg, int Z, int MO,
                             DLStreamHandle stream_handle = NULL) {
    size_t size = ArrSize(grad_values);
    size_t roarsz = ArrSize(param);
    int blk = grad_indices->shape[(grad_indices->ndim) - 1];
    size_t length = (grad_values->shape[(grad_values->ndim) - 1]);

    const float *grad_data = (const float *)grad_values->data;
    float *param_data = (float *)param->data;
    const int *indices_data = (const int *)grad_indices->data;
    const int *x_data = (const int *)grad_x->data;
/*
    printf("%d\n",roarsz);

    printf("%d\n",grad_indices->ndim);
    for (int i=0;i<(grad_indices->ndim);++i)
        printf("%d ",(grad_indices->shape[i]));
    printf("\n");

    printf("%d\n",grad_values->ndim);
    for (int i=0;i<(grad_values->ndim);++i)
        printf("%d ",(grad_values->shape[i]));
    printf("\n");

    printf("%d\n",grad_x->ndim);
    for (int i=0;i<(grad_x->ndim);++i)
        printf("%d ",(grad_x->shape[i]));
    printf("\n");
*/

    dim3 blocks;
    dim3 threads;
    ThreadBlock1D(threads, blocks, size);


    if (stream_handle ){
        sgd_robe_update<<<blocks, threads, 0,
                            *(hipStream_t *)stream_handle->handle>>>(
            grad_data, indices_data, x_data, param_data, size, length, lr, roarsz, Bg, Cg, Dg,Z,blk,MO);
    }
    else{
        sgd_robe_update<<<blocks, threads>>>(grad_data, indices_data, x_data,
                                               param_data, size, length, lr, roarsz, Bg, Cg, Dg,Z,blk,MO);
    }
    
    return 0;
}
