#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void less_const_kernel(float *input, float *output, float threshold,
                                  size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = (input[ind] < threshold);
}

int DLGpuNumLessThan(const DLArrayHandle input, DLArrayHandle middle,
                     DLArrayHandle output, float threshold, int *axes,
                     int num_ax, DLStreamHandle stream_handle = NULL) {
    size_t size = ArrSize(input);
    float *input_data = (float *)input->data;
    float *middle_data = (float *)middle->data;
    dim3 blocks;
    dim3 threads;
    ThreadBlock1D(threads, blocks, size);
    if (stream_handle)
        less_const_kernel<<<blocks, threads, 0,
                            *(hipStream_t *)stream_handle->handle>>>(
            input_data, middle_data, threshold, size);
    else
        less_const_kernel<<<blocks, threads>>>(input_data, middle_data,
                                               threshold, size);
    return DLGpuReduceSum(middle, output, axes, num_ax, stream_handle);
}

__global__ void set_less_const_kernel(float *arr, float threshold,
                                      size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    if (abs(arr[ind]) < threshold) {
        arr[ind] = 0;
    }
}

int DLGpuSetLessThan(const DLArrayHandle arr, float threshold,
                     DLStreamHandle stream_handle = NULL) {
    size_t size = ArrSize(arr);
    float *arr_data = (float *)arr->data;
    dim3 blocks;
    dim3 threads;
    ThreadBlock1D(threads, blocks, size);
    if (stream_handle)
        set_less_const_kernel<<<blocks, threads, 0,
                                *(hipStream_t *)stream_handle->handle>>>(
            arr_data, threshold, size);
    else
        set_less_const_kernel<<<blocks, threads>>>(arr_data, threshold, size);
    return 0;
}

__global__ void get_larger_than_kernel_feature_dimension(const float *arr,
                                                         const float *threshold,
                                                         int *mask,
                                                         size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    mask[ind] = (abs(arr[ind]) > threshold[ind]);
}

__global__ void get_larger_than_kernel_feature(const float *arr,
                                               const float *threshold,
                                               int *mask, size_t size,
                                               size_t dim) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    size_t rind = ind / dim;
    mask[ind] = (abs(arr[ind]) > threshold[rind]);
}

__global__ void get_larger_than_kernel_dimension(const float *arr,
                                                 const float *threshold,
                                                 int *mask, size_t size,
                                                 size_t dim) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    size_t cind = ind % dim;
    mask[ind] = (abs(arr[ind]) > threshold[cind]);
}

__global__ void get_larger_than_kernel_global(const float *arr,
                                              const float *threshold, int *mask,
                                              size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    mask[ind] = (abs(arr[ind]) > threshold[0]);
}

int DLGpuGetLargerThan(const DLArrayHandle input, const DLArrayHandle threshold,
                       DLArrayHandle mask,
                       DLStreamHandle stream_handle = NULL) {
    assert(input->ndim == 2);
    size_t size = ArrSize(input);
    size_t dim = input->shape[1];
    const float *input_data = (const float *)input->data;
    const float *thres_data = (const float *)threshold->data;
    int *mask_data = (int *)mask->data;
    size_t thres_ndim = threshold->ndim;
    size_t last_dim = threshold->shape[thres_ndim - 1];
    bool use_feature = (thres_ndim > 1);
    bool use_dimension = (last_dim > 1);
    dim3 blocks;
    dim3 threads;
    ThreadBlock1D(threads, blocks, size);
    assert(stream_handle != NULL);
    hipStream_t stream = *(hipStream_t *)stream_handle->handle;
    if (use_feature && use_dimension) {
        get_larger_than_kernel_feature_dimension<<<blocks, threads, 0,
                                                   stream>>>(
            input_data, thres_data, mask_data, size);
    } else if (!use_feature && use_dimension) {
        get_larger_than_kernel_dimension<<<blocks, threads, 0, stream>>>(
            input_data, thres_data, mask_data, size, dim);
    } else if (use_feature && !use_dimension) {
        get_larger_than_kernel_feature<<<blocks, threads, 0, stream>>>(
            input_data, thres_data, mask_data, size, dim);
    } else {
        get_larger_than_kernel_global<<<blocks, threads, 0, stream>>>(
            input_data, thres_data, mask_data, size);
    }
    return 0;
}
