#include "hip/hip_runtime.h"
#include "gpu_runtime.h"

__global__ void embedding_lookup_kernel(const float *input, const int *ids,
                                        float *output, size_t size,
                                        size_t length, size_t input_row) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size)
        return;
    int id = ids[index];
    float *output_ptr = output + length * index;
    if (id < 0 || id >= input_row) {
        for (int i = 0; i < length; i++)
            output_ptr[i] = 0;
    } else {
        const float *input_ptr = input + length * id;
        for (int i = 0; i < length; i++)
            output_ptr[i] = input_ptr[i];
    }
}

int DLGpuEmbeddingLookUp(const DLArrayHandle input, const DLArrayHandle ids,
                         DLArrayHandle output,
                         DLStreamHandle stream_handle = NULL) {
    assert(input->ndim == 2);
    size_t size = 1;
    for (int i = 0; i < output->ndim; i++) {
        if (i < output->ndim - 1) {
            assert(ids->shape[i] == output->shape[i]);
        } else if (i == output->ndim - 1) {
            assert(input->shape[1] == output->shape[i]);
        }
    }
    for (int i = 0; i < ids->ndim; i++) {
        size = size * ids->shape[i];
    }
    size_t input_row = input->shape[0];
    size_t length = input->shape[1];
    dim3 blocks;
    dim3 threads;
    float *output_data = (float *)output->data;
    const float *input_data = (const float *)input->data;
    const int *id_list = (const int *)ids->data;
    if (size <= 1024) {
        threads.x = size;
        blocks.x = 1;
    } else {
        threads.x = 1024;
        blocks.x = (size + 1023) / 1024;
    }
    if (stream_handle)
        embedding_lookup_kernel<<<blocks, threads, 0,
                                  *(hipStream_t *)stream_handle->handle>>>(
            input_data, id_list, output_data, size, length, input_row);
    else
        embedding_lookup_kernel<<<blocks, threads>>>(
            input_data, id_list, output_data, size, length, input_row);
    return 0;
}

__global__ void array_set_zero_kernel(float *output, size_t size) {
    size_t ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind >= size)
        return;
    output[ind] = 0;
}
